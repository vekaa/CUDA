#include <hip/hip_runtime.h>
#include <cstdlib>


#include <iostream>
#include <ctime>
#include <fstream>
using namespace std;

__global__ void MatrixMul(int *a, int *b, int *c, int N) {
	//Calculate the global row and colomn for each thread
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;


	if (row < N && col < N) {
		int temp = 0;
		for (int i = 0; i < N; i++) {
			temp += a[row * N + i] * b[i * N + col];
		}

		//Write back the resolt
		c[row * N + col] = temp;
	}
}

void CreateMatrix(int *a, int N) {
	for (int i = 0; i < N * N; i++)
		a[i] = rand() % 100;
}

clock_t Test(int N) {
	
	size_t bytes = N * N * sizeof(int);
	clock_t clocks;

	//Alocate memory
	int* a, * b, * c;

	hipMallocManaged(&a, bytes);
	hipMallocManaged(&b, bytes);
	hipMallocManaged(&c, bytes);

	//Create matrix
	CreateMatrix(a, N);
	CreateMatrix(b, N);


	//Set CTA and grid dimension
	int threads = 16;
	int blocks = (N + threads - 1) / threads;

	dim3 THREADS(threads, threads);
	dim3 BLOCKS(blocks, blocks);

	clocks = clock();
	MatrixMul <<<BLOCKS, THREADS >>> (a, b, c, N);
	hipDeviceSynchronize();
	clocks = clock() - clocks;

	cout << "Program completed successfully " << endl;
	cout << "Dimension: "<< N << " Clocks: " << clocks << " Time: " << (double)clocks / CLOCKS_PER_SEC << endl;

	hipFree(a);
	hipFree(b);
	hipFree(c);

	return clocks;
}

int main() {
	int i = 0;
	cout << " PRogram starting " << i << endl;
	clock_t clk;
	ofstream resolt;
	resolt.open("resolt.txt");

	while (i <= 10000) {
		clk = Test(i);
		resolt << i << " " << clk << " " << (double)clk / CLOCKS_PER_SEC << endl;
		if (i < 100)
			i += 10;
		else if (i < 1000)
			i += 100;
		else
			i += 1000;
	}
	resolt.close();
	return 0;
}